#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>

__global__ void copy_kernel(int *d_in, int *d_out, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = idx; i < N; i += blockDim.x * gridDim.x)
    d_out[i] = d_in[i];
}

int main(int argc, char **argv) {
  uint SIZE = 512;

  int *A, *B;
  int *dA, *dB;

  A = (int *)malloc(sizeof(int) * SIZE);
  B = (int *)malloc(sizeof(int) * SIZE);

  hipMalloc((void **)&dA, sizeof(int) * SIZE);
  hipMalloc((void **)&dB, sizeof(int) * SIZE);

  for (int i = 0; i < SIZE; i++) {
    A[i] = i;
  }

  hipMemcpy(dA, A, sizeof(float) * SIZE, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float) * SIZE, hipMemcpyHostToDevice);

  int threads = 128;
  int blocks = 4;
  copy_kernel<<<blocks, threads>>>(dA, dB, SIZE);

  hipMemcpy(B, dB, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

  free(A);
  free(B);
  hipFree(dA);
  hipFree(dB);

  return 0;
}
