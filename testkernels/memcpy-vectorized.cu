#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void copy_kernel ( int* d_in, int* d_out, int N ) { 
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = idx; i < N/4; i += blockDim.x * gridDim.x) {
    reinterpret_cast<int4*>(d_out)[i] = reinterpret_cast<int4*>(d_in)[i];
  }

  // in only one thread, process final elements (if there are any)
  int remainder = N%4;
  if (idx==N/4 && remainder!=0) {
    while(remainder) {
      int idx = N - remainder--;
      d_out[idx] = d_in[idx];
    }
  }
}

int main(int argc, char **argv) {
    uint SIZE = 512;

    int *A, *B;
    int *dA, *dB;

    A = (int *) malloc(sizeof(int) * SIZE);
    B = (int *) malloc(sizeof(int) * SIZE);

    hipMalloc((void **) &dA, sizeof(int) * SIZE);
    hipMalloc((void **) &dB, sizeof(int) * SIZE);

    for (int i = 0; i < SIZE; i++) {
        A[i] = i;
    }

    hipMemcpy(dA, A, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    int threads = 128; 
    int blocks = 1;  
    copy_kernel<<<blocks, threads>>>(dA, dB, SIZE); 

    hipMemcpy(B, dB, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

    free(A);
    free(B);
    hipFree(dA);
    hipFree(dB);

    return 0;
}
