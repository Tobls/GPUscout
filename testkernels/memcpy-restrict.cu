#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void copy_kernel ( int *__restrict__ d_in, int *__restrict__ d_out, int N ) { 
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) 
        d_out[i] = d_in[i]; 
}

int main(int argc, char **argv) {
    uint SIZE = 512;

    int *A, *B;
    int *dA, *dB;

    A = (int *) malloc(sizeof(int) * SIZE);
    B = (int *) malloc(sizeof(int) * SIZE);

    hipMalloc((void **) &dA, sizeof(int) * SIZE);
    hipMalloc((void **) &dB, sizeof(int) * SIZE);

    for (int i = 0; i < SIZE; i++) {
        A[i] = i;
    }

    hipMemcpy(dA, A, sizeof(float) * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * SIZE, hipMemcpyHostToDevice);

    int threads = 128; 
    int blocks = 4;  
    copy_kernel<<<blocks, threads>>>(dA, dB, SIZE); 

    hipMemcpy(B, dB, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

    free(A);
    free(B);
    hipFree(dA);
    hipFree(dB);

    return 0;
}
