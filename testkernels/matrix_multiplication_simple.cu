#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void kernel(int *a, int* b, int* c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int sum = 0; 
    if( col < k && row < m) {  
        for(int i = 0; i < n; i++) { 
            sum += a[row * n + i] * b[i * k + col]; 
        } 
        c[row * k + col] = sum; 
    }  
}

int main(int argc, char **argv) {
    uint SIZE = 32;

    int *A, *B, *C;
    int *dA, *dB, *dC;

    A = (int *) malloc(sizeof(int) * SIZE * SIZE);
    B = (int *) malloc(sizeof(int) * SIZE * SIZE);
    C = (int *) malloc(sizeof(int) * SIZE * SIZE);

    hipMalloc((void **) &dA, sizeof(int) * SIZE * SIZE);
    hipMalloc((void **) &dB, sizeof(int) * SIZE * SIZE);
    hipMalloc((void **) &dC, sizeof(int) * SIZE * SIZE);

    for (int i = 0; i < SIZE * SIZE; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
        C[i] = 0.0f;
    }

    hipMemcpy(dA, A, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);

    dim3 gridDim(1, 1, 1);
    dim3 blockDim(32, 32, 1);

    kernel<<<gridDim, blockDim>>>(dA, dB, dC, SIZE, SIZE, SIZE);

    hipMemcpy(C, dC, sizeof(int) * SIZE * SIZE, hipMemcpyDeviceToHost);

    free(A);
    free(B);
    free(C);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}
