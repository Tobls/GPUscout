#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>

__global__ void axpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a*x[i] + y[i];
    }
}

int main(int argc, char **argv) {
  uint SIZE = 512;

  float *A, *B;
  float *dA, *dB;

  A = (float *)malloc(sizeof(float) * SIZE);
  B = (float *)malloc(sizeof(float) * SIZE);

  hipMalloc((void **)&dA, sizeof(float) * SIZE);
  hipMalloc((void **)&dB, sizeof(float) * SIZE);

  for (int i = 0; i < SIZE; i++) {
    A[i] = i;
  }

  hipMemcpy(dA, A, sizeof(float) * SIZE, hipMemcpyHostToDevice);
  hipMemcpy(dB, B, sizeof(float) * SIZE, hipMemcpyHostToDevice);

  int threads = 128;
  int blocks = 4;
  axpy<<<blocks, threads>>>(SIZE, 5, dA, dB);

  hipMemcpy(B, dB, sizeof(float) * SIZE, hipMemcpyDeviceToHost);

  free(A);
  free(B);
  hipFree(dA);
  hipFree(dB);

  return 0;
}
